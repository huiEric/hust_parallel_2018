#include "hip/hip_runtime.h"
// ��� CUDA ��
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <cstdlib>
#include<opencv2/opencv.hpp>
#include <time.h>
#include<iostream>

#define PARTS 4

using namespace cv;
using namespace std;

typedef struct st_range {
	int x1;
	int y1;
	int x2;
	int y2;
}StRange;

Mat srcImage, grayImage, binarygray;

const int N = 100;
const int BLOCK_data = 1;	// ����
const int THREAD_data = 4;	// �����е��߳���

static void * g_binary(void *range)		//��ֵ��
{
	StRange rg = *(StRange *)range;
	binarygray = Mat::zeros(grayImage.rows, grayImage.cols, grayImage.type());
	for (int i = rg.x1; i < rg.x2; i++)
	{
		for (int j = rg.y1; j < rg.y2; j++)
		{
			if (grayImage.data[i*grayImage.step + j] > 128)
			{
				binarygray.data[i*binarygray.step + j] = 255;		//white
			}
			else
			{
				binarygray.data[i*binarygray.step + j] = 0;			//black
			}
		}
	}
	return NULL;
}
// �˺����������˵��ã��豸��ִ�С�
__global__ static void g_dilation(unsigned char *imgData, unsigned char *result, int rows, int cols)  //��ʴ
{
	StRange srcRange, rg, ranges[PARTS];
	srcRange = { 0, 0, rows, cols };
	//�з�ͼ��
	ranges[0] = { 0, 0, srcRange.x2 / 4, srcRange.y2 };
	ranges[1] = { srcRange.x2 / 4, 0, srcRange.x2 / 2, srcRange.y2 };
	ranges[2] = { srcRange.x2 / 2 ,0 , 3 * srcRange.x2 / 4, srcRange.y2 };
	ranges[3] = { 3 * srcRange.x2 / 4, 0 , srcRange.x2, srcRange.y2 };
	for (int tid = 0; tid < 4; tid++)
	{
		if (tid == threadIdx.x)
		{
			//printf("thid:%d\n", tid);
			rg = ranges[tid];
			//printf("x1: %d, y1: %d\nx2: %d, y2:%d\n", ranges[tid].x1, ranges[tid].y1, ranges[tid].x2, ranges[tid].y2);
			for (int i = rg.x1; i < rg.x2; i++)
			{
				for (int j = rg.y1; j < rg.y2; j++)
				{
					if (imgData[(i - 1)*cols + j] + imgData[(i - 1)*cols + j + 1] + imgData[i*cols + j + 1] == 0)
					{
						result[i*cols + j] = 0;
					}
					else
					{
						result[i*cols + j] = 255;
					}
				}
			}
			//printf("Over thread%d\n\n", tid);
		}
	}
}
__global__ static void g_erosion(unsigned char *imgData, unsigned char *result, int rows, int cols)  //����
{
	StRange srcRange, rg, ranges[PARTS];
	srcRange = { 0, 0, rows, cols };
	//�з�ͼ��
	ranges[0] = { 0, 0, srcRange.x2 / 4, srcRange.y2 };
	ranges[1] = { srcRange.x2 / 4, 0, srcRange.x2 / 2, srcRange.y2 };
	ranges[2] = { srcRange.x2 / 2 ,0 , 3 * srcRange.x2 / 4, srcRange.y2 };
	ranges[3] = { 3 * srcRange.x2 / 4, 0 , srcRange.x2, srcRange.y2 };
	for (int tid = 0; tid < 4; tid++)
	{
		if (tid == threadIdx.x)
		{
			printf("thid:%d\n", tid);
			rg = ranges[tid];
			//printf("x1: %d, y1: %d\nx2: %d, y2:%d\n", ranges[tid].x1, ranges[tid].y1, ranges[tid].x2, ranges[tid].y2);
			for (int i = rg.x1; i < rg.x2; i++)
			{
				for (int j = rg.y1; j < rg.y2; j++)
				{
					if (imgData[(i - 1)*cols + j] == 0 || imgData[(i - 1)*cols + j - 1] == 0 || imgData[i*cols + j + 1] == 0)
					{
						result[i*cols + j] = 0;
					}
					else
					{
						result[i*cols + j] = 255;
					}
				}
			}
			printf("Over thread%d\n\n", tid);
		}
	}
}
// CUDA��ʼ������
bool InitCUDA()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);	// ��ȡ��ʾ�豸��
	if (deviceCount == 0)
	{
		cout << "�Ҳ����豸" << endl;
		return EXIT_FAILURE;
	}
	int i;
	for (i = 0; i<deviceCount; i++)
	{
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) // ��ȡ�豸����
		{
			if (prop.major >= 1) //cuda��������
			{
				break;
			}
		}
	}
	if (i == deviceCount)
	{
		cout << "�Ҳ���֧�� CUDA ������豸" << endl;
		return EXIT_FAILURE;
	}
	hipSetDevice(i); // ѡ��ʹ�õ���ʾ�豸
	return EXIT_SUCCESS;
}

int main()
{
	if (InitCUDA()) // ��ʼ�� CUDA ���뻷��
		return EXIT_FAILURE;
	cout << "�ɹ����� CUDA ���㻷��" << endl << endl;

	clock_t begin, end;
	double cost;
	cout << "\n\n�������漰����" << "��ʴ��erosion�������ͣ�dilation)��\n\n";
	
	system("color 3f");
	srcImage = imread("D://2.jpg");
	imshow("ԭͼ", srcImage);
	cvtColor(srcImage, grayImage, CV_RGB2GRAY);		//RGBͼ��ת��Ϊ�Ҷ�ͼ
	StRange srcRange = { 0, 0, srcImage.rows, srcImage.cols };
	g_binary(&srcRange);		//�Ҷ�ͼ��ֵ������
	imshow("binarygray", binarygray);

	// ���ݲ���
	unsigned char *img, *result1, *result2;
	int arraySize = sizeof(unsigned char)*srcImage.cols * srcImage.rows;

	//��ʼ��¼
	begin = clock();
	// ���Դ���Ϊ������󿪱ٿռ�
	hipMalloc((void**)&img, arraySize);
	// ���Դ���Ϊ������󿪱ٿռ�
	hipMalloc((void**)&result1, arraySize);
	hipMalloc((void**)&result2, arraySize);

	// �����ݴ�����Դ�
	hipMemcpy(img, binarygray.data, arraySize, hipMemcpyHostToDevice);

	// ���� kernel ���� - �˺������Ը����Դ��ַ�Լ�����Ŀ�ţ��̺߳Ŵ������ݡ�
	g_erosion << <BLOCK_data, THREAD_data, 0 >> > (img, result1, srcImage.rows, srcImage.cols);
	g_dilation << <BLOCK_data, THREAD_data, 0 >> > (img, result2, srcImage.rows, srcImage.cols);
	// ���ڴ���Ϊ������󿪱ٿռ�
	unsigned char * resData1 = new unsigned char[srcImage.rows * srcImage.cols];
	unsigned char * resData2 = new unsigned char[srcImage.rows * srcImage.cols];
	// ���Դ��ȡ����Ľ��
	hipMemcpy(resData1, result1, arraySize, hipMemcpyDeviceToHost);
	hipMemcpy(resData2, result2, arraySize, hipMemcpyDeviceToHost);

	Mat erosion(srcImage.rows, srcImage.cols, CV_8UC1, resData1);
	Mat dilation(srcImage.rows, srcImage.cols, CV_8UC1, resData2);
	imshow("dilation", dilation);
	imshow("erosion", erosion);

	// �ͷ��Դ�
	hipFree(img);
	hipFree(result1);
	hipFree(result2);

	end = clock();
	cost = (double)(end - begin);
	printf("Time cost is: %lf ms", cost);
	waitKey(0);
	return 0;
}